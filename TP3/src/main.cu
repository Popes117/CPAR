#include "hip/hip_runtime.h"
#include "EventManager.h"
#include "fluid_solver.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#define SIZE 168

#define IX(i, j, k) ((i) + (M + 2) * (j) + (M + 2) * (N + 2) * (k))

#define NUM_BLOCKS 512
#define NUM_THREADS_PER_BLOCK 256
#define TOTALSIZE NUM_BLOCKS*NUM_THREADS_PER_BLOCK

// Globals for the grid size
static int M = SIZE;
static int N = SIZE;
static int O = SIZE;
static float dt = 0.1f;      // Time delta
static float diff = 0.0001f; // Diffusion constant
static float visc = 0.0001f; // Viscosity constant

// Fluid simulation arrays
static float *u, *v, *w, *u_prev, *v_prev, *w_prev;
static float *dens, *dens_prev;
float *du, *dv, *dw;
float *du_prev, *dv_prev, *dw_prev;
float *ddens, *ddens_prev;

// Function to allocate simulation data
int allocate_data() {
  int size = (M + 2) * (N + 2) * (O + 2);
  int bytes = size * sizeof(float);
  u = new float[size];
  v = new float[size];
  w = new float[size];
  u_prev = new float[size];
  v_prev = new float[size];
  w_prev = new float[size];
  dens = new float[size];
  dens_prev = new float[size];
  hipMalloc((void **)&du, bytes);
  hipMalloc((void **)&dv, bytes);
  hipMalloc((void **)&dw, bytes);
  hipMalloc((void **)&du_prev, bytes);
  hipMalloc((void **)&dv_prev, bytes);
  hipMalloc((void **)&dw_prev, bytes);
  hipMalloc((void **)&ddens, bytes);
  hipMalloc((void **)&ddens_prev, bytes);

  if (!u || !v || !w || !u_prev || !v_prev || !w_prev || !dens || !dens_prev) {
    std::cerr << "Cannot allocate memory" << std::endl;
    return 0;
  }
  return 1;
}

// Function to clear the data (set all to zero)
void clear_data() {
  int size = (M + 2) * (N + 2) * (O + 2);
  for (int i = 0; i < size; i++) {
    u[i] = v[i] = w[i] = u_prev[i] = v_prev[i] = w_prev[i] = dens[i] =
        dens_prev[i] = 0.0f;
  }
  hipMemcpy(du, u, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dv, v, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dw, w, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(du_prev, u_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dv_prev, v_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dw_prev, w_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(ddens, dens, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(ddens_prev, dens_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
}

// Free allocated memory
void free_data() {
  delete[] u;
  delete[] v;
  delete[] w;
  delete[] u_prev;
  delete[] v_prev;
  delete[] w_prev;
  delete[] dens;
  delete[] dens_prev;
  hipFree(du);
  hipFree(dv);
  hipFree(dw);
  hipFree(du_prev);
  hipFree(dv_prev);
  hipFree(dw_prev);
  hipFree(ddens);
  hipFree(ddens_prev);
}


__global__ void apply_events_kernel(Event *events, int num_events, int center_idx, float *du, float *dv, float *dw, float *ddens) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Cada thread trata de um evento
    if (idx < num_events) {
        Event event = events[idx];

        if (event.type == ADD_SOURCE) {
            // Aplicar densidade no centro
            ddens[center_idx] = event.density;
        } else if (event.type == APPLY_FORCE) {
            // Aplicar forças no centro
            du[center_idx] = event.force.x;
            dv[center_idx] = event.force.y;
            dw[center_idx] = event.force.z;
        }
    }
}


// Apply events (source or force) for the current timestep
void apply_events(const std::vector<Event> &events,int idx, float *dens, float *u, float *v, float *w) {

  int size = events.size();
  Event *d_events;
  hipMalloc((void **)&d_events, size * sizeof(Event));
  hipMemcpy(d_events, events.data(), size * sizeof(Event), hipMemcpyHostToDevice);
  int threads_per_block = 256;
  int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;

  // Lançar o kernel
  apply_events_kernel<<<blocks_per_grid, threads_per_block>>>(d_events, size, idx, u, v, w, dens);

  hipFree(d_events);

}

#if 0

template <unsigned int blockSize>
__global__ void reduce_sum_density(float *g_idata, float *g_odata, unsigned int n) {
    extern __shared__ float sdata[];  // Memória compartilhada
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + tid;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    // Inicializa memória compartilhada
    sdata[tid] = 0;

    // Soma os elementos atribuídos ao thread
    while (i < n) {
        sdata[tid] += g_idata[i];
        if (i + blockSize < n) {
            sdata[tid] += g_idata[i + blockSize];
        }
        i += gridSize;  // Incrementa para processar elementos restantes
    }
    __syncthreads();

    // Redução em memória compartilhada
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32) {
        volatile float *vshared = sdata; // Evita leitura de memória global
        vshared[tid] += vshared[tid + 32];
        vshared[tid] += vshared[tid + 16];
        vshared[tid] += vshared[tid + 8];
        vshared[tid] += vshared[tid + 4];
        vshared[tid] += vshared[tid + 2];
        vshared[tid] += vshared[tid + 1];
    }

    // Escreve o resultado parcial na memória global
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

float sum_density(float *ddens, int size) {
    const int threadsPerBlock = 512;  // Ajustável dependendo do hardware
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Alocar memória para resultados intermediários
    float *d_intermediate;
    hipMalloc(&d_intermediate, blocksPerGrid * sizeof(float));

    // Soma total
    float total_density = 0.0f;

    // Primeira chamada ao kernel
    reduce_sum_density<512><<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(float)>>>(ddens, d_intermediate, size);

    // Reduzir iterativamente até restar um único bloco
    while (blocksPerGrid > 1) {
        int newBlocksPerGrid = (blocksPerGrid + threadsPerBlock - 1) / threadsPerBlock;
        reduce_sum_density<512><<<newBlocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(float)>>>(d_intermediate, d_intermediate, blocksPerGrid);
        blocksPerGrid = newBlocksPerGrid;
    }

    // Copiar o resultado final para o host
    hipMemcpy(&total_density, d_intermediate, sizeof(float), hipMemcpyDeviceToHost);

    // Liberar memória
    hipFree(d_intermediate);

    return total_density;
}

#else

//Function to sum the total density
float sum_density() {
  hipMemcpy(dens, ddens, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyDeviceToHost);
  float total_density = 0.0f;
  int size = (M + 2) * (N + 2) * (O + 2);
  for (int i = 0; i < size; i++) {
    total_density += dens[i];
  }
  return total_density;
}
#endif 
// Simulation loop
void simulate(EventManager &eventManager, int timesteps) {
  int i = M / 2, j = N / 2, k = O / 2;
  int idx = IX(i, j, k);

  for (int t = 0; t < timesteps; t++) {
    // Get the events for the current timestep
    std::vector<Event> events = eventManager.get_events_at_timestamp(t);

    // Apply events to the simulation
    apply_events(events,idx, ddens, du, dv, dw);

    // Perform the simulation steps
    vel_step(M, N, O, du, dv, dw, du_prev, dv_prev, dw_prev, visc, dt);
    dens_step(M, N, O, ddens, ddens_prev, du, dv, dw, diff, dt);
    std::cout << "Timestep " << t << std::endl;
  }
}

int main() {
  // Initialize EventManager
  EventManager eventManager;
  eventManager.read_events("events.txt");

  // Get the total number of timesteps from the event file
  int timesteps = eventManager.get_total_timesteps();

  // Allocate and clear data
  if (!allocate_data())
    return -1;
  clear_data();

  // Run simulation with events
  simulate(eventManager, timesteps);

  // Print total density at the end of simulation
  //float total_density = sum_density(ddens, (M + 2) * (N + 2) * (O + 2));
  float total_density = sum_density();
  std::cout << "Total density after " << timesteps
            << " timesteps: " << total_density << std::endl;

  // Free memory
  free_data();

  return 0;
}