#include "hip/hip_runtime.h"
#include "EventManager.h"
#include "fluid_solver.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#define SIZE 168

#define IX(i, j, k) ((i) + (M + 2) * (j) + (M + 2) * (N + 2) * (k))

// Globals for the grid size
static int M = SIZE;
static int N = SIZE;
static int O = SIZE;
static float dt = 0.1f;      // Time delta
static float diff = 0.0001f; // Diffusion constant
static float visc = 0.0001f; // Viscosity constant

// Fluid simulation arrays
static float *u, *v, *w, *u_prev, *v_prev, *w_prev;
static float *dens, *dens_prev;
float *du, *dv, *dw;
float *du_prev, *dv_prev, *dw_prev;
float *ddens, *ddens_prev;
float *changes_d, *d_max_c, *d_intermediate;

// Function to allocate simulation data
int allocate_data() {
  int size = (M + 2) * (N + 2) * (O + 2);
  int bytes = size * sizeof(float);
  const unsigned int blockSize = 128; 
  const unsigned int gridSize = (size + blockSize * 2 - 1) / (blockSize * 2);
  u = new float[size];
  v = new float[size];
  w = new float[size];
  u_prev = new float[size];
  v_prev = new float[size];
  w_prev = new float[size];
  dens = new float[size];
  dens_prev = new float[size];
  hipMalloc((void **)&du, bytes);
  hipMalloc((void **)&dv, bytes);
  hipMalloc((void **)&dw, bytes);
  hipMalloc((void **)&du_prev, bytes);
  hipMalloc((void **)&dv_prev, bytes);
  hipMalloc((void **)&dw_prev, bytes);
  hipMalloc((void **)&ddens, bytes);
  hipMalloc((void **)&ddens_prev, bytes);
  hipMalloc((void **)&changes_d, bytes);
  hipMalloc((void **)&d_max_c, sizeof(float));
  hipMalloc((void **)&d_intermediate, gridSize * sizeof(float));


  if (!u || !v || !w || !u_prev || !v_prev || !w_prev || !dens || !dens_prev) {
    std::cerr << "Cannot allocate memory" << std::endl;
    return 0;
  }
  return 1;
}

// Function to clear the data (set all to zero)
void clear_data() {
  int size = (M + 2) * (N + 2) * (O + 2);
  for (int i = 0; i < size; i++) {
    u[i] = v[i] = w[i] = u_prev[i] = v_prev[i] = w_prev[i] = dens[i] =
        dens_prev[i] = 0.0f;
  }
  hipMemcpy(du, u, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dv, v, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dw, w, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(du_prev, u_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dv_prev, v_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dw_prev, w_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(ddens, dens, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(ddens_prev, dens_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
}

// Free allocated memory
void free_data() {
  delete[] u;
  delete[] v;
  delete[] w;
  delete[] u_prev;
  delete[] v_prev;
  delete[] w_prev;
  delete[] dens;
  delete[] dens_prev;
  hipFree(du);
  hipFree(dv);
  hipFree(dw);
  hipFree(du_prev);
  hipFree(dv_prev);
  hipFree(dw_prev);
  hipFree(ddens);
  hipFree(ddens_prev);
  hipFree(changes_d);
  hipFree(d_max_c);
  hipFree(d_intermediate);
}

__global__ void apply_events_kernel(const Event *events, int num_events, float *u, float *v, float *w, float *dens, int idx) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int e = 0; e < num_events; e++) {
            Event event = events[e];
            if (event.type == ADD_SOURCE) {
                dens[idx] = event.density;
            } else if (event.type == APPLY_FORCE) {
                u[idx] = event.force.x;
                v[idx] = event.force.y;
                w[idx] = event.force.z;
            }
        }
    }
}

// Função principal para gerenciar transferência e execução
void apply_events(const std::vector<Event> &events, int idx, float *dens, float *u, float *v, float *w) {
  int size = events.size();
  if (size == 0) return;
  Event *d_events;
  hipMalloc((void **)&d_events, size * sizeof(Event));
  hipMemcpy(d_events, events.data(), size * sizeof(Event), hipMemcpyHostToDevice);

  // Lançar o kernel
  apply_events_kernel<<<1, 1>>>(d_events, size, u, v, w, dens, idx);

  hipFree(d_events);
}

//Function to sum the total density
float sum_density() {
  hipMemcpy(dens, ddens, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyDeviceToHost);
  float total_density = 0.0f;
  int size = (M + 2) * (N + 2) * (O + 2);
  for (int i = 0; i < size; i++) {
    total_density += dens[i];
  }
  return total_density;
}

// Simulation loop
void simulate(EventManager &eventManager, int timesteps) {
  int i = M / 2, j = N / 2, k = O / 2;
  int idx = IX(i, j, k);

  for (int t = 0; t < timesteps; t++) {
    // Get the events for the current timestep
    std::vector<Event> events = eventManager.get_events_at_timestamp(t);

    // Apply events to the simulation
    apply_events(events,idx, ddens, du, dv, dw);

    // Perform the simulation steps
    vel_step(M, N, O, du, dv, dw, du_prev, dv_prev, dw_prev, visc, dt, changes_d, d_max_c, d_intermediate);
    dens_step(M, N, O, ddens, ddens_prev, du, dv, dw, diff, dt, changes_d, d_max_c, d_intermediate);
  }
}

int main() {
  // Initialize EventManager
  EventManager eventManager;
  eventManager.read_events("events.txt");

  // Get the total number of timesteps from the event file
  int timesteps = eventManager.get_total_timesteps();

  // Allocate and clear data
  if (!allocate_data())
    return -1;
  clear_data();

  // Run simulation with events
  simulate(eventManager, timesteps);

  // Print total density at the end of simulation
  float total_density = sum_density();
  std::cout << "Total density after " << timesteps
            << " timesteps: " << total_density << std::endl;

  // Free memory
  free_data();

  return 0;
}