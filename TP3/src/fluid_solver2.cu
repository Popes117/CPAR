#include "hip/hip_runtime.h"
#include "fluid_solver.h"
#include <cmath>
#include <algorithm>
#include <iostream>
#include <omp.h>
#include <hip/hip_runtime.h>

#define IX(i, j, k) ((i) + (val) * (j) + (val) * (val2) * (k))  //Compute 1 dimensional (1D) index from 3D coordinates
#define SWAP(x0, x){float *tmp = x0;x0 = x;x = tmp;}            //Swap two pointers
#define MAX(a, b) (((a) > (b)) ? (a) : (b))                     //Get maximum between two values
#define LINEARSOLVERTIMES 20                                    //Number of iterations for the linear solver

//Global values to minimize the number of calculations of the index between steps
int ix000, ix100, ix010, ix001;
int ixm100, ixm00, ixm110, ixm101;
int ix0n10, ix1n10, ix0n0, ix0n11;
int ixm1n10, ixmn10, ixm1n0, ixm1n11;

__global__ void add_source_kernel(int M, int N, int O, float *x, float *s, float dt) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // O tamanho total da grade
    int size = (M + 2) * (N + 2) * (O + 2);

    // Garantir que o índice não ultrapasse o tamanho da grade
    if (idx < size) {
        x[idx] += dt * s[idx];
    }
}

void launch_add_source_kernel(int M, int N, int O, float *x, float *s, float dt) {
    int size = (M + 2) * (N + 2) * (O + 2);

    // Configuração de threads e blocos
    int threadsPerBlock = 1024;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Chamada ao kernel
    add_source_kernel<<<blocksPerGrid, threadsPerBlock>>>(M, N, O, x, s, dt);

    // Sincronizar para garantir que o kernel tenha concluído a execução
    hipDeviceSynchronize();

    //hipError_t err = hipGetLastError();
    //if (err != hipSuccess) {
    //    printf("CUDA error after add_source_kernel launch: %s\n", hipGetErrorString(err));
    //}
}

__global__ void set_bnd_kernel(
    int M, int N, int O, int b, float *x,
    int ix010, int ix001, int ix100, int ix000,
    int ixm110, int ixm101, int ixm00, int ixm100,
    int ix0n0, int ix0n11, int ix1n10, int ix0n10,
    int ixm1n0, int ixm1n11, int ixmn10, int ixm1n10
){

    int i = threadIdx.x + blockIdx.x * blockDim.x; // thread index in x
    int j = threadIdx.y + blockIdx.y * blockDim.y; // thread index in y
    
    int val = M + 2;
    int val2 = N + 2;

    if (i >= M || j >= N || i < 0 || j < 0) return; // Ensure within bounds

    float neg_mask;

    // Handle boundaries for b == 3 (z-axis faces)
    if (b == 3) {
        neg_mask = -1.0f;
        int index = IX(0, j + 1, 0);
        int first_index = IX(0, j + 1, 1);
        int last_index = IX(0, j + 1, O);
        int idx = IX(0, j + 1, O + 1);

        if (i < M) {
            const auto first_value = x[first_index + i];
            const auto last_value = x[last_index + i];
            x[index + i] = neg_mask * first_value;
            x[idx + i] = neg_mask * last_value;
        }
    }

    // Handle boundaries for b == 1 (x-axis faces)
    if (b == 1) {
        neg_mask = -1.0f;
        int index = IX(0, j + 1, 0);
        int first_index = IX(1, j + 1, 0);
        int last_index = IX(M, j + 1, 0);
        int idx = IX(M + 1, j + 1, 0);

        if (i < M) {
            const auto first_value = x[first_index + i];
            const auto last_value = x[last_index + i];
            x[index + i] = neg_mask * first_value;
            x[idx] = neg_mask * last_value;
        }
    }

    // Handle boundaries for b == 2 (y-axis faces)
    if (b == 2) {
        neg_mask = -1.0f;
        int index = IX(i + 1, 0, 0);
        int first_index = IX(i + 1, 1, 0);
        int last_index = IX(i + 1, N, 0);
        int idx = IX(i + 1, N + 1, 0);

        if (i < M) {
            const auto first_value = x[first_index + i];
            const auto last_value = x[last_index + i];
            x[index + i] = neg_mask * first_value;
            x[idx] = neg_mask * last_value;
        }
    }

    // Handle corners (only one thread does this)
    if (i == 0 && j == 0) {
        x[ix000] = 0.33f * (x[ix100] + x[ix010] + x[ix001]);
        x[ixm100] = 0.33f * (x[ixm00] + x[ixm110] + x[ixm101]);
        x[ix0n10] = 0.33f * (x[ix1n10] + x[ix0n0] + x[ix0n11]);
        x[ixm1n10] = 0.33f * (x[ixmn10] + x[ixm1n0] + x[ixm1n11]);
    }
}

void launch_set_bnd_kernel(int M, int N, int O, int b, float *x) {

    dim3 blockDim(16, 16);
    dim3 gridDim((M + blockDim.x - 1) / blockDim.x,
                   (N + blockDim.y - 1) / blockDim.y);
    set_bnd_kernel<<<gridDim, blockDim>>>(
        M, N, O, b, x,
        ix010, ix001, ix100, ix000,
        ixm110, ixm101, ixm00, ixm100,
        ix0n0, ix0n11, ix1n10, ix0n10,
        ixm1n0, ixm1n11, ixmn10, ixm1n10
    );
   
}   

__device__ float atomicMaxFloat(float *address, float value) {
    int *address_as_int = (int *)address; // Reinterpreta o endereço como inteiro
    int old = *address_as_int, assumed;

    do {
        assumed = old;
        // Usa __int_as_float para comparar os valores como floats
        old = atomicCAS(address_as_int, assumed, 
                        __float_as_int(fmaxf(value, __int_as_float(assumed))));
    } while (assumed != old);

    return __int_as_float(old); // Retorna o valor máximo final como float
}

__device__ void atomicMaxFloatPrecise(float *address, float val) {
    float old = *address;  // Lê o valor atual
    while (val > old) {    // Continua enquanto o novo valor for maior
        float assumed = old;
        old = atomicCAS((int *)address, __float_as_int(assumed), __float_as_int(val));
    }
}

__global__ void lin_solve_kernel(int M, int N, int O, int b, float *x, float *x0, float a, float c, bool process_red, float *changes) {
    
    int val = M + 2;
    int val2 = N + 2;
    float divv = 1.0f / c;
    int y = M + 2;
    int z = (M + 2) * (N + 2);
    int color = int(process_red);

    // Índices globais baseados em thread e bloco
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int i = 2 * (blockIdx.x * blockDim.x + threadIdx.x) + 1 + (j + k + color) % 2 ; // Garante que começa em 1

    // Verifica se está dentro dos limites
    if (i > M || j > N || k > O) return;

    int idx = IX(i, j, k);
    float old_x = x[idx];

    // Atualiza o valor de x[idx] com a fórmula dada
    x[idx] = (x0[idx] +
              a * (x[idx - 1] + x[idx + 1] +
                   x[idx - y] + x[idx + y] +
                   x[idx - z] + x[idx + z])) * divv;

    // Calcula a alteração e atualiza max_c de forma atômica
    changes[idx] = fabsf(x[idx] - old_x);
}


void lin_solve_kernel(int M, int N, int O, int b, float *x, float *x0, float a, float c) {
    float tol = 1e-7, max_c;
    int l = 0;
    int size = (M + 2) * (N + 2) * (O + 2) * sizeof(float);
    float *changes_d;
    hipMalloc(&changes_d, size);
    // Aloca `max_c` em memória global no device

    // Configuração do grid e blocos
    dim3 blockDim(16, 16, 4);
    // Para a maneira do Artur, divide o M/2
    dim3 gridDim((M/2 + blockDim.x - 1) / blockDim.x,
                 (N + blockDim.y - 1) / blockDim.y,
                 (O + blockDim.z - 1) / blockDim.z);

    do {
        max_c = 0.0f;
        hipMemset(changes_d, 0, size);

        // Processa células pretas
        lin_solve_kernel<<<gridDim, blockDim>>>(M, N, O, b, x, x0, a, c, false, changes_d);
        //hipDeviceSynchronize();

        hipError_t err1 = hipGetLastError();
        if (err1 != hipSuccess) {
            printf("CUDA error depois do lin solve 1 : %s\n", hipGetErrorString(err1));
        }

        // Processa células vermelhas
        lin_solve_kernel<<<gridDim, blockDim>>>(M, N, O, b, x, x0, a, c, true, changes_d);
        //hipDeviceSynchronize();
        hipError_t err2 = hipGetLastError();
        if (err2 != hipSuccess) {
            printf("CUDA error depois do lin solve 2 : %s\n", hipGetErrorString(err2));
        }

        // TODO : Kernel que verifica o máximo do array `changes_d` e atualiza `max_c`

        // Atualiza os limites com o kernel `set_bnd`
        launch_set_bnd_kernel(M, N, O, b, x);

    } while (++l < 20);

    // Libera a memória alocada para `d_max_c`
    hipFree(changes_d);
}

// Diffusion step (uses implicit method)
void diffuse(int M, int N, int O, int b, float *x, float *x0, float diff, float dt) {
    int max = MAX(MAX(M, N), O);
    float a = dt * diff * max * max;
    int size = (M + 2) * (N + 2) * (O + 2) * sizeof(float);

    //float *copy = (float *)malloc(size);
    //hipMemcpy(copy, x, size, hipMemcpyDeviceToHost);

    lin_solve_kernel(M, N, O, b, x, x0, a, 1 + 6 * a);
    //hipDeviceSynchronize();

    //hipError_t err = hipGetLastError();
    //if (err != hipSuccess) {
    //    printf("CUDA error : %s\n", hipGetErrorString(err));
    //}

    //float *x_h = (float *)malloc(size);
    //hipMemcpy(x_h, x, size, hipMemcpyDeviceToHost);

    // Copy x to host
    //bool is_different = false;
    //for (int idx = 0; idx < (M + 2) * (N + 2) * (O + 2); idx++) {
    //    if (x_h[idx] != copy[idx]) {
    //        is_different = true;
    //        break;
    //    }
    //}

    // Print the result
    //if (is_different){
    //    printf("x is different from the copy after lin_solve.\n");
    //}
    //else {
    //    printf("x is identical to the copy after lin_solve.\n");
    //}

}

__global__ void advect_kernel(int M, int N, int O, int b, float *d, float *d0, float *u, float *v, float *w, float dt) {

    float dtX = dt * M, dtY = dt * N, dtZ = dt * O;
    int val = M + 2;
    int val2 = N + 2;

    int i = blockIdx.x * blockDim.x + threadIdx.x + 1; // Índice em x
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1; // Índice em y
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1; // Índice em z

    if (i > M || j > N || k > O) return; // Fora dos limites

    int idx = IX(i, j, k);

    // Calcula as posições retroativas
    float x = i - dtX * u[idx];
    float y = j - dtY * v[idx];
    float z = k - dtZ * w[idx];

    // Clamping para garantir que esteja dentro do domínio
    x = fminf(fmaxf(x, 0.5f), M + 0.5f);
    y = fminf(fmaxf(y, 0.5f), N + 0.5f);
    z = fminf(fmaxf(z, 0.5f), O + 0.5f);

    // Índices inteiros para interpolação
    int i0 = (int)x, i1 = i0 + 1;
    int j0 = (int)y, j1 = j0 + 1;
    int k0 = (int)z, k1 = k0 + 1;

    // Pesos de interpolação
    float s1 = x - i0, s0 = 1 - s1;
    float t1 = y - j0, t0 = 1 - t1;
    float u1 = z - k0, u0 = 1 - u1;

    // Recuperar valores para interpolação
    float d0_i0j0k0 = d0[IX(i0, j0, k0)];
    float d0_i0j0k1 = d0[IX(i0, j0, k1)];
    float d0_i0j1k0 = d0[IX(i0, j1, k0)];
    float d0_i0j1k1 = d0[IX(i0, j1, k1)];
    float d0_i1j0k0 = d0[IX(i1, j0, k0)];
    float d0_i1j0k1 = d0[IX(i1, j0, k1)];
    float d0_i1j1k0 = d0[IX(i1, j1, k0)];
    float d0_i1j1k1 = d0[IX(i1, j1, k1)];

    // Interpolação 3D
    d[idx] = s0 * (t0 * (u0 * d0_i0j0k0 + u1 * d0_i0j0k1) +
                   t1 * (u0 * d0_i0j1k0 + u1 * d0_i0j1k1)) +
             s1 * (t0 * (u0 * d0_i1j0k0 + u1 * d0_i1j0k1) +
                   t1 * (u0 * d0_i1j1k0 + u1 * d0_i1j1k1));

}

void launch_advect_kernel(int M, int N, int O, int b, float *d, float *d0, float *u, float *v, float *w, float dt) {

    dim3 blockDim(16, 16, 4);
    dim3 gridDim((M + 2 + blockDim.x - 1) / blockDim.x,
                   (N + 2 + blockDim.y - 1) / blockDim.y,
                   (O + 2 + blockDim.z - 1) / blockDim.z);
    advect_kernel<<<gridDim,blockDim>>>(M, N, O, b, d, d0, u, v, w, dt);
    launch_set_bnd_kernel(M, N, O, b, d);
    
}   


// Advection step (uses velocity field to move quantities)
void advect(int M, int N, int O, int b, float *d, float *d0, float *u, float *v, float *w, float dt) {
    float dtX = dt * M, dtY = dt * N, dtZ = dt * O;

    launch_advect_kernel(M, N, O, b, d, d0, u, v, w, dt);  
    launch_set_bnd_kernel(M, N, O, b, d);

}

__global__ void loop1_project_kernel(int M, int N, int O, float *u, float *v, float *w, float *p, float *div) {
    int val = M + 2;
    int val2 = N + 2;
    int y = M + 2;
    int z = (M + 2) * (N + 2); 
    int max = MAX(M, MAX(N, O));
    float invMax = 1.0f / max;

    int i = blockIdx.x * blockDim.x + threadIdx.x + 1; // Índice em x
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1; // Índice em y
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1; // Índice em z

    if (i < 1 || j < 1 || k < 1 || i > M || j > N || k > O) return;

    int idx = IX(i, j, k);

    div[idx] = (-0.5f * (u[idx + 1] - u[idx - 1] + v[idx + y] -
                         v[idx - y] + w[idx + z] - w[idx - z])) * invMax;
    p[idx] = 0.0f;
}

__global__ void loop2_project_kernel(int M, int N, int O, float *u, float *v, float *w, float *p) {
    int val = M + 2;
    int val2 = N + 2;
    int y = M + 2;
    int z = (M + 2) * (N + 2); 
    int max = MAX(M, MAX(N, O));
    float invMax = 1.0f / max;

    int i = blockIdx.x * blockDim.x + threadIdx.x + 1; // Índice em x
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1; // Índice em y
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1; // Índice em z

    // Adjustment of u, v, and w without loop blocking

    if (i < 1 || j < 1 || k < 1 || i > M || j > N || k > O) return;

    int idx = IX(i, j, k);

    u[idx] -= 0.5f * (p[idx + 1] - p[idx - 1]);
    v[idx] -= 0.5f * (p[idx + y] - p[idx - y]);
    w[idx] -= 0.5f * (p[idx + z] - p[idx - z]);
}

void project(int M, int N, int O, float *u, float *v, float *w, float *p, float *divv) {
    int max = MAX(M, MAX(N, O));
    float invMax = 1.0f / max;

    dim3 blockDim(16, 16, 4);
    dim3 gridDim((M + blockDim.x - 1) / blockDim.x,
                   (N + blockDim.y - 1) / blockDim.y,
                   (O + blockDim.z - 1) / blockDim.z);
    loop1_project_kernel<<<gridDim, blockDim>>>(M, N, O, u, v, w, p, divv);
    hipDeviceSynchronize();

    launch_set_bnd_kernel(M, N, O, 0, divv);
    launch_set_bnd_kernel(M, N, O, 0, p);
    lin_solve_kernel(M, N, O, 0, p, divv, 1, 6);
    //hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error : %s\n", hipGetErrorString(err));
    }

    loop2_project_kernel<<<gridDim, blockDim>>>(M, N, O, u, v, w, p);
    hipDeviceSynchronize();

    launch_set_bnd_kernel(M, N, O, 1, u),
    launch_set_bnd_kernel(M, N, O, 2, v);
    launch_set_bnd_kernel(M, N, O, 3, w);

}

// Step function for density
void dens_step(int M, int N, int O, float *x, float *x0, float *u, float *v, float *w, float diff, float dt) {  

  launch_add_source_kernel(M, N, O, x, x0, dt); 
  SWAP(x0, x);
  diffuse(M, N, O, 0, x, x0, diff, dt);
  SWAP(x0, x);
  advect(M, N, O, 0, x, x0, u, v, w, dt);

}

// Step function for velocity
void vel_step(int M, int N, int O, float *u, float *v, float *w, float *u0, float *v0, float *w0, float visc, float dt) {
  // Define global values
  int val = M + 2;
  int val2 = N + 2;
  ix000 = IX(0, 0, 0);
  ix100 = IX(1, 0, 0);
  ix010 = IX(0, 1, 0);
  ix001 = IX(0, 0, 1);
  ixm100 = IX(M + 1, 0, 0);
  ixm00 = IX(M, 0, 0);
  ixm110 = IX(M + 1, 1, 0);
  ixm101 = IX(M + 1, 0, 1);
  ix0n10 = IX(0, N + 1, 0);
  ix1n10 = IX(1, N + 1, 0);
  ix0n0 = IX(0, N, 0);
  ix0n11 = IX(0, N + 1, 1);
  ixm1n10 = IX(M + 1, N, 0);
  ixmn10 = IX(M, N + 1, 0);
  ixm1n0 = IX(M + 1, N, 0);
  ixm1n11 = IX(M + 1, N + 1, 1);

  launch_add_source_kernel(M, N, O, u, u0, dt);
  launch_add_source_kernel(M, N, O, v, v0, dt);
  launch_add_source_kernel(M, N, O, w, w0, dt);

  SWAP(u0, u);
  diffuse(M, N, O, 1, u, u0, visc, dt);
  SWAP(v0, v);
  diffuse(M, N, O, 2, v, v0, visc, dt);
  SWAP(w0, w);
  diffuse(M, N, O, 3, w, w0, visc, dt);
  project(M, N, O, u, v, w, u0, v0);
  SWAP(u0, u);
  SWAP(v0, v);
  SWAP(w0, w);
  advect(M, N, O, 1, u, u0, u0, v0, w0, dt);
  //if (err != hipSuccess) {
  //    printf("CUDA error : %s\n", hipGetErrorString(err));
  //}
  advect(M, N, O, 2, v, v0, u0, v0, w0, dt);
  advect(M, N, O, 3, w, w0, u0, v0, w0, dt);
  project(M, N, O, u, v, w, u0, v0);
}
